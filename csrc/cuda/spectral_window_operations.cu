#include "hip/hip_runtime.h"


#define PI_THIRD 1.047197551196597746154214461093167628065723133125f
#define WINDOW_ADJOINT_PARAM(N,m) (PI_THIRD * m / (N*N))


// The Fourier coefficients of the actual window function phi_act are
//   phi_act_hat(k) = 1/M * exp(-b pi^2 / M^2 k^2).
// Because we need to take the inverse and multiply with M later on, we
// instead evaluate
//   phi_hat_inv(k) = 1 / (M phi_hat_act(k)) = exp(b_square_pi_over_M * k^2)
// with b_square_pi_over_M = b * (pi / M)^2.
// In our case with oversampling rate 2:
//   b_square_pi_over_M = (4 * m * pi) / (3 * M^2) = (m * pi) / (3 * N^2)
__device__ __forceinline__ float
eval_phi_hat_inv(const int64_t k, const float window_b_square_pi_over_M)
{
    return expf(float(k*k) * window_b_square_pi_over_M);
}



// Fill array phi_hat_inv with the inverse N-periodic Fourier coefficients of the Gaussian window function,
//   phi_hat_inv[freq_idx] = 1/(M*phi_act_hat(freq_idx))
// for all freq_idx in [0,...,N/2].
// Because of symmetry, we can later obtain the remaining freq_idx in [-N/2,...,1] via
//   1/phi_hat(freq_idx) = 1/phi_hat(-freq_idx) = phi_hat_inv[-freq_idx].
__global__ void
compute_phi_hat_inv_kernel(
    float* phi_hat_inv,
    const int64_t halfN,
    const float window_b_square_pi_over_M)
{
    int64_t freq_idx;

    for (freq_idx=blockDim.x*blockIdx.x + threadIdx.x; freq_idx <= halfN; freq_idx += gridDim.x*blockDim.x)
    {
        phi_hat_inv[freq_idx] = eval_phi_hat_inv(freq_idx, window_b_square_pi_over_M);

#ifdef NFFT_PRINT_DEBUG
        printf(" - phi_hat_inv at frequency %ld:  %f\n", freq_idx, phi_hat_inv[freq_idx]);
#endif
    }
}



// Fill tensor y such that for all batch_idx, column_idx, and
//   freq_idx = ((i[0]*N + i[1])*N + ...)*N + i[dim-1],
//   (referring to the actual frequency multiindex l with l[d] = i[d] - N/2)
// the output is stored in y[batch_idx][freq_idx][column_idx]
__global__ void
complex_adjoint_rolloff_correction_kernel(
    torch::PackedTensorAccessor64<c10::complex<float>,3> y_acc, // size batch_size x N^dim x num_columns
    const hipfftComplex *g_hat, // size batch_size * num_columns * M^dim
    const float *phi_hat_inv, // size N/2 + 1
    const int64_t dim, const int64_t batch_size, const int64_t num_columns,
    const int64_t N, const int64_t halfN, const int64_t prod_N)
{
    int64_t batch_idx, freq_idx, reverse_freq_idx, f, column_idx, g_hat_idx, d;
    float factor;

    for (batch_idx=blockIdx.z*blockDim.z + threadIdx.z; batch_idx < batch_size; batch_idx += gridDim.z*blockDim.z)
    {
        for (column_idx = blockIdx.y*blockDim.y + threadIdx.y; column_idx < num_columns; column_idx += gridDim.y*blockDim.y)
        {
            for (reverse_freq_idx = blockIdx.x*blockDim.x + threadIdx.x; reverse_freq_idx < prod_N; reverse_freq_idx += gridDim.x*blockDim.x)
            {
                factor = 1.0f;
                // we actually iterate over the frequency indices in reverse order:
                //   reverse_freq_idx = ((i[dim-1]*N + i[dim-2])*N + ...)*N + i[0]
                // so we can obtain the current i[d] as f % N if we always shave off f /= N in each iteration.
                f = reverse_freq_idx;
                // But for the index in y, we need to also build the original freq_idx
                freq_idx = 0;
                // The correct index can be computed iteratively:
                //   g_hat_idx = ((((batch_idx*num_columns + column_idx)*M + i[0])*M + i[0])*M + ...)*(N+1) * i[dim-1])
                g_hat_idx = batch_idx*num_columns + column_idx;
                for (d=0; d<dim; ++d)
                {
                    freq_idx = freq_idx*N + (f % N);
                    if (f % N < halfN) {
                        // first half: actual frequency (f % N) - halfN is negative
                        // g_hat value is stored in the end part
                        g_hat_idx = g_hat_idx*2*N + 2*N + (f % N) - halfN;
                        // phi_hat value is stored at absolute value of actual frequency
                        factor *= phi_hat_inv[halfN - (f % N)];
                    }
                    else {
                        // second half: actual frequency (f % N) - halfN is non-negative
                        // g_hat value is stored in the first part
                        g_hat_idx = g_hat_idx*2*N + (f % N) - halfN;
                        // phi_hat value is stored at actual frequency
                        factor *= phi_hat_inv[(f % N) - halfN];
                    }
                    f /= N;
                }

                y_acc[batch_idx][freq_idx][column_idx] =
                    c10::complex<float>(hipCrealf(g_hat[g_hat_idx])*factor, hipCimagf(g_hat[g_hat_idx])*factor);

#ifdef NFFT_PRINT_DEBUG
                if (batch_idx == 0 && column_idx == 0)
                    printf(" - Output at freq index %ld = [%ld, %ld, %ld], oversampled frequency %ld = [%ld, %ld, %ld]:  g_hat=%f + %fi, factor=%f\n",
                            freq_idx, reverse_freq_idx % N, (reverse_freq_idx / N) % N, reverse_freq_idx / (N * N),
                            g_hat_idx, g_hat_idx / (4*N*N), (g_hat_idx / (2*N)) % (2*N), g_hat_idx % (2*N),
                            hipCrealf(g_hat[g_hat_idx]), hipCimagf(g_hat[g_hat_idx]), factor);
#endif
            }
        }
    }
}

// real version of complex_adjoint_rolloff_correction_kernel
__global__ void
real_adjoint_rolloff_correction_kernel(
    torch::PackedTensorAccessor64<float,3> y_acc, // size batch_size x N^dim x num_columns
    const hipfftComplex *g_hat, // size batch_size * num_columns * M^dim
    const float *phi_hat_inv, // size N/2 + 1
    const int64_t dim, const int64_t batch_size, const int64_t num_columns,
    const int64_t N, const int64_t halfN, const int64_t prod_N)
{
    int64_t batch_idx, freq_idx, reverse_freq_idx, f, column_idx, g_hat_idx, d;
    float factor;

    for (batch_idx=blockIdx.z*blockDim.z + threadIdx.z; batch_idx < batch_size; batch_idx += gridDim.z*blockDim.z)
    {
        for (column_idx = blockIdx.y*blockDim.y + threadIdx.y; column_idx < num_columns; column_idx += gridDim.y*blockDim.y)
        {
            for (reverse_freq_idx = blockIdx.x*blockDim.x + threadIdx.x; reverse_freq_idx < prod_N; reverse_freq_idx += gridDim.x*blockDim.x)
            {
                factor = 1.0f;
                f = reverse_freq_idx;
                freq_idx = 0;
                g_hat_idx = batch_idx*num_columns + column_idx;
                for (d=0; d<dim; ++d)
                {
                    freq_idx = freq_idx*N + (f % N);
                    if (f % N < halfN) {
                        g_hat_idx = g_hat_idx*2*N + 2*N + (f % N) - halfN;
                        factor *= phi_hat_inv[halfN - (f % N)];
                    }
                    else {
                        g_hat_idx = g_hat_idx*2*N + (f % N) - halfN;
                        factor *= phi_hat_inv[(f % N) - halfN];
                    }
                    f /= N;
                }

                y_acc[batch_idx][freq_idx][column_idx] = hipCrealf(g_hat[g_hat_idx])*factor;
            }
        }
    }
}




__global__ void
complex_forward_rolloff_correction_kernel(
    const torch::PackedTensorAccessor64<c10::complex<float>,3> x_acc, // size batch_size x prod_N x num_columns
    hipfftComplex *g_hat, // size batch_size * num_columns * prod_M
    const float *phi_hat_inv, // size N/2 + 1
    const int64_t dim, const int64_t batch_size, const int64_t num_columns,
    const int64_t N, const int64_t halfN, const int64_t prod_N)
{
    int64_t batch_idx, freq_idx, reverse_freq_idx, f, column_idx, g_hat_idx, d;
    float factor;

    for (batch_idx=blockIdx.z*blockDim.z + threadIdx.z; batch_idx < batch_size; batch_idx += gridDim.z*blockDim.z)
    {
        for (column_idx = blockIdx.y*blockDim.y + threadIdx.y; column_idx < num_columns; column_idx += gridDim.y*blockDim.y)
        {
            for (reverse_freq_idx = blockIdx.x*blockDim.x + threadIdx.x; reverse_freq_idx < prod_N; reverse_freq_idx += gridDim.x*blockDim.x)
            {
                factor = 1.0f;
                // we actually iterate over the frequency indices in reverse order:
                //   reverse_freq_idx = ((i[dim-1]*N + i[dim-2])*N + ...)*N + i[0]
                // so we can obtain the current i[d] as f % N if we always shave off f /= N in each iteration.
                // Here i[d] in [0,N-1] corresponds to the frequency i[d]-N/2 in [-N/2,N/2-1].
                f = reverse_freq_idx;
                // But for the index in x, we need to also build the original
                //   freq_idx = ((i[0]*N + i[1])*N + ...)*N + i[dim-1]
                freq_idx = 0;
                // The correct index can be computed iteratively:
                //   g_hat_idx = ((((batch_idx*num_columns + column_idx)*M + i[0])*M + i[0])*M + ...)*(N+1) * i[dim-1])
                g_hat_idx = batch_idx*num_columns + column_idx;
                for (d=0; d<dim; ++d)
                {
                    freq_idx = freq_idx*N + (f % N);
                    if (f % N < halfN) {
                        // first half: actual frequency (f % N) - halfN is negative
                        // g_hat value is stored in the end part
                        g_hat_idx = g_hat_idx*2*N + 2*N + (f % N) - halfN;
                        // phi_hat value is stored at absolute value of actual frequency
                        factor *= phi_hat_inv[halfN - (f % N)];
                    }
                    else {
                        // second half: actual frequency (f % N) - halfN is non-negative
                        // g_hat value is stored in the first part
                        g_hat_idx = g_hat_idx*2*N + (f % N) - halfN;
                        // phi_hat value is stored at actual frequency
                        factor *= phi_hat_inv[(f % N) - halfN];
                    }
                    f /= N;
                }


                g_hat[g_hat_idx] = make_hipFloatComplex(x_acc[batch_idx][freq_idx][column_idx].real() * factor,
                                                        x_acc[batch_idx][freq_idx][column_idx].imag() * factor);

#ifdef NFFT_PRINT_DEBUG
                if (batch_idx == 0 && column_idx == 0)
                    printf(" - g_hat in oversampled frequency [%ld, %ld, %ld], original frequency [%ld, %ld, %ld]:  g_hat=%f + %fi, factor=%f\n",
                            g_hat_idx / (2*N*(N+1)), (g_hat_idx / (N+1)) % (2*N), g_hat_idx % (N+1),
                            reverse_freq_idx % N, (reverse_freq_idx / N) % N, reverse_freq_idx / (N * N),
                            hipCrealf(g_hat[g_hat_idx]), hipCimagf(g_hat[g_hat_idx]), factor);
#endif
            }
        }
    }
}


// Variant of complex_forward_rolloff_correction_kernel for real-valued tensor x
__global__ void
real_forward_rolloff_correction_kernel(
    const torch::PackedTensorAccessor64<float,3> x_acc, // size batch_size x prod_N x num_columns
    hipfftComplex *g_hat, // size batch_size * num_columns * prod_M
    const float *phi_hat_inv, // size N/2 + 1
    const int64_t dim, const int64_t batch_size, const int64_t num_columns,
    const int64_t N, const int64_t halfN, const int64_t prod_N)
{
    int64_t batch_idx, freq_idx, reverse_freq_idx, f, column_idx, g_hat_idx, d;
    float factor;

    for (batch_idx=blockIdx.z*blockDim.z + threadIdx.z; batch_idx < batch_size; batch_idx += gridDim.z*blockDim.z)
    {
        for (column_idx = blockIdx.y*blockDim.y + threadIdx.y; column_idx < num_columns; column_idx += gridDim.y*blockDim.y)
        {
            for (reverse_freq_idx = blockIdx.x*blockDim.x + threadIdx.x; reverse_freq_idx < prod_N; reverse_freq_idx += gridDim.x*blockDim.x)
            {
                factor = 1.0f;
                f = reverse_freq_idx;
                freq_idx = 0;
                g_hat_idx = batch_idx*num_columns + column_idx;
                for (d=0; d<dim; ++d)
                {
                    freq_idx = freq_idx*N + (f % N);
                    if (f % N < halfN) {
                        g_hat_idx = g_hat_idx*2*N + 2*N + (f % N) - halfN;
                        factor *= phi_hat_inv[halfN - (f % N)];
                    }
                    else {
                        g_hat_idx = g_hat_idx*2*N + (f % N) - halfN;
                        factor *= phi_hat_inv[(f % N) - halfN];
                    }
                    f /= N;
                }


                g_hat[g_hat_idx] = make_hipFloatComplex(x_acc[batch_idx][freq_idx][column_idx] * factor, 0.0f);
            }
        }
    }
}



__global__ void
complex_kernel_convolution_kernel(
    const torch::PackedTensorAccessor64<c10::complex<float>,1> coeffs_acc,
    hipfftComplex *g_hat, // size batch_size * num_columns * prod_M
    const float *phi_hat_inv, // size N/2 + 1
    const int64_t dim, const int64_t batch_size, const int64_t num_columns,
    const int64_t M, const int64_t halfN, const int64_t prod_M)
{
    int64_t batch_idx, column_idx, reverse_freq_idx, coeff_idx, g_hat_idx, f, d;
    float factor;
    c10::complex<float> coeff;

    for (batch_idx=blockIdx.z*blockDim.z + threadIdx.z; batch_idx < batch_size; batch_idx += gridDim.z*blockDim.z)
    {
        for (column_idx = blockIdx.y*blockDim.y + threadIdx.y; column_idx < num_columns; column_idx += gridDim.y*blockDim.y)
        {
            for (reverse_freq_idx = blockIdx.x*blockDim.x + threadIdx.x; reverse_freq_idx < prod_M; reverse_freq_idx += gridDim.x*blockDim.x)
            {
                factor = 1.0f;
                coeff_idx = 0;
                g_hat_idx = batch_idx*num_columns + column_idx;

                for (d=0; d<dim; ++d) {
                    // a coeff_idx below zero indicates that this part of g_hat
                    // is set to zero and there is no fitting coefficient
                    if (coeff_idx >= 0) {
                        if (f % M < halfN) {
                            // First quarter of g_hat: refers to positive actual
                            // frequency (f % M) in [0,...,halfN-1]
                            // coefficient is stored in second half of coeffs
                            coeff_idx = 2*halfN*coeff_idx + halfN + (f % M);
                            // phi_hat value is stored in the actual frequency index
                            factor *= phi_hat_inv[f % M];
                        }
                        else if (f % M >= 3*halfN) {
                            // Last quarter of g_hat: refers to negative actual
                            // frequency (f % M) - M in [-halfN,...,-1]
                            // coefficient stored in first half of coeffs
                            coeff_idx = 2*halfN*coeff_idx + (f % M) - 3*halfN;
                            // phi_hat value is stored in the absolute of the actual
                            // frequency index, M - (f % M) in [1,...,halfN]
                            factor *= phi_hat_inv[M - (f % M)];
                        }
                        else {
                            coeff_idx = -1;
                        }
                    }

                    g_hat_idx = M*g_hat_idx + (f % M);
                }

                if (coeff_idx < 0) {
                    g_hat[g_hat_idx] = make_hipFloatComplex(0.0f, 0.0f);
                }
                else {
                    coeff = coeffs_acc[coeff_idx];
                    g_hat[g_hat_idx] = make_hipFloatComplex(
                        factor * (hipCrealf(g_hat[g_hat_idx])*coeff.real() - hipCimagf(g_hat[g_hat_idx])*coeff.imag()),
                        factor * (hipCrealf(g_hat[g_hat_idx])*coeff.imag() + hipCimagf(g_hat[g_hat_idx])*coeff.real()));
                }
            }
        }
    }
}


__global__ void
real_kernel_convolution_kernel(
    const torch::PackedTensorAccessor64<float,1> coeffs_acc,
    hipfftComplex *g_hat, // size batch_size * num_columns * prod_M
    const float *phi_hat_inv, // size N/2 + 1
    const int64_t dim, const int64_t batch_size, const int64_t num_columns,
    const int64_t M, const int64_t halfN, const int64_t prod_M)
{
    int64_t batch_idx, column_idx, reverse_freq_idx, coeff_idx, g_hat_idx, f, d;
    float factor;

    for (batch_idx=blockIdx.z*blockDim.z + threadIdx.z; batch_idx < batch_size; batch_idx += gridDim.z*blockDim.z)
    {
        for (column_idx = blockIdx.y*blockDim.y + threadIdx.y; column_idx < num_columns; column_idx += gridDim.y*blockDim.y)
        {
            for (reverse_freq_idx = blockIdx.x*blockDim.x + threadIdx.x; reverse_freq_idx < prod_M; reverse_freq_idx += gridDim.x*blockDim.x)
            {
                factor = 1.0f;
                coeff_idx = 0;
                g_hat_idx = batch_idx*num_columns + column_idx;

                for (d=0; d<dim; ++d) {
                    // a coeff_idx below zero indicates that this part of g_hat
                    // is set to zero and there is no fitting coefficient
                    if (coeff_idx >= 0) {
                        if (f % M < halfN) {
                            // First quarter of g_hat: refers to positive actual
                            // frequency (f % (2*N)) in [0,...,halfN-1]
                            // coefficient is stored in second half of coeffs
                            coeff_idx = 2*halfN*coeff_idx + halfN + (f % M);
                            // phi_hat value is stored in the actual frequency index
                            factor *= phi_hat_inv[f % M];
                        }
                        else if (f % M >= 3*halfN) {
                            // Last quarter of g_hat: refers to negative actual
                            // frequency (f % (2*N)) - 2*N in [-halfN,...,-1]
                            // coefficient stored in first half of coeffs
                            coeff_idx = 2*halfN*coeff_idx + (f % M) - 3*halfN;
                            // phi_hat value is stored in the absolute of the actual
                            // frequency index, 2*N - (f % (2*N)) in [1,...,halfN]
                            factor *= phi_hat_inv[M - (f % M)];
                        }
                        else {
                            coeff_idx = -1;
                        }
                    }

                    g_hat_idx = M*g_hat_idx + (f % M);
                }

                if (coeff_idx < 0) {
                    g_hat[g_hat_idx] = make_hipFloatComplex(0.0f, 0.0f);
                }
                else {
                    factor *= coeffs_acc[coeff_idx];
                    g_hat[g_hat_idx] = make_hipFloatComplex(
                        factor * hipCrealf(g_hat[g_hat_idx]),
                        factor * hipCimagf(g_hat[g_hat_idx]));
                }
            }
        }
    }
}
